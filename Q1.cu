
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

__global__ void sumArray(float *X, float *result, int size) {
    __shared__ float partialSum[BLOCK_SIZE];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    /*initialize partial sum for this thread*/
    float sum = 0.0f;
    if (index < size) {
        sum = X[index];
    }
    
    partialSum[tid] = sum;
    __syncthreads();

    // Perform reduction to compute the final sum
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && index + stride < size) {
            partialSum[tid] += partialSum[tid + stride];
        }
        __syncthreads();
    }

    /*write the final sum to global memory*/
    if (tid == 0) {
        result[blockIdx.x] = partialSum[0];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <input_file>\n", argv[0]);
        return 1;
    }

    char *input_file = argv[1];

    FILE *file = fopen(input_file, "r");
    if (file == NULL) {
        printf("Error: Unable to open file %s.\n", input_file);
        return 1;
    }

    /*count the number of elements in the file*/
    int size = 0;
    float temp;
    while (fscanf(file, "%f", &temp) == 1) {
        size++;
    }
    fseek(file, 0, SEEK_SET); /*reset file pointer to the beginning of the file*/

    float *h_arr = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++) {
        fscanf(file, "%f", &h_arr[i]);
    }
    fclose(file);

    float *d_arr;
    float *d_result;

    /*Allocate device memory for array and result*/
    hipMalloc((void **)&d_arr, size * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    /*copy array to device*/
    hipMemcpy(d_arr, h_arr, size * sizeof(float), hipMemcpyHostToDevice);

    /*Launch kernel*/
    sumArray<<<1, BLOCK_SIZE>>>(d_arr, d_result, size);

    /*copy result back to host*/
    float h_result;
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    
    printf("%f\n", h_result);

    /*Free device memory*/
    hipFree(d_arr);
    hipFree(d_result);

    /*Free host memory*/
    free(h_arr);

    return 0;
}
